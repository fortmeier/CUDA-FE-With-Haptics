#include "hip/hip_runtime.h"
/***************************************************************************/
/*                       CUDA based TLED Solver                            */
/*                     {c} 2008-2010 Karsten Noe                           */
/*                      The Alexandra Institute                            */
/*                   See our blog on cg.alexandra.dk                       */ 
/***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "TetrahedralMesh.h"


__global__ void
precalculateShapeFunctionDerivatives_k(ShapeFunctionDerivatives *shape_function_derivatives, Tetrahedron *tetrahedra, Point *points, unsigned int numTets)
{
	int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (me_idx>=numTets)
		return;

	ShapeFunctionDerivatives sfd;
	
	Tetrahedron tet = tetrahedra[me_idx];

	if (tet.x<0) return; // illegal tetrahedron

	float4 a = points[tet.x];
	float4 b = points[tet.y];
	float4 c = points[tet.z];
	float4 d = points[tet.w];

	float denominator = c.y*d.x*b.z + a.x*c.y*d.z + a.y*d.x*c.z - c.x*b.y*a.z + c.x*d.y*a.z - a.x*b.y*d.z +
		a.x*b.y*c.z + c.x*b.y*d.z - c.x*a.y*d.z + a.y*c.x*b.z - a.y*d.x*b.z - d.x*b.y*c.z + b.x*c.y*a.z + 
		b.x*d.y*c.z - b.x*a.y*c.z - b.x*c.y*d.z - b.x*d.y*a.z + b.x*a.y*d.z - d.y*c.x*b.z + d.y*a.x*b.z -
		d.y*a.x*c.z - c.y*a.x*b.z - d.x*c.y*a.z + d.x*b.y*a.z;

	sfd.h1.x = (c.y*d.z - b.y*d.z + b.y*c.z + d.y*b.z - d.y*c.z - c.y*b.z)/denominator;
	sfd.h1.y = -(-c.x*b.z + d.x*b.z + c.x*d.z - b.x*d.z + b.x*c.z - d.x*c.z)/denominator;
	sfd.h1.z = (-c.x*b.y + c.x*d.y - b.x*d.y + b.x*c.y - d.x*c.y + d.x*b.y)/denominator;

	sfd.h2.x = -(c.y*d.z - c.y*a.z + d.y*a.z - a.y*d.z + c.z*a.y - d.y*c.z)/denominator;
	sfd.h2.y = (c.x*d.z - a.x*d.z - c.x*a.z + d.x*a.z - d.x*c.z + a.x*c.z)/denominator;
	sfd.h2.z = -(-a.x*d.y + a.x*c.y + d.x*a.y - c.x*a.y + c.x*d.y - d.x*c.y)/denominator;

	sfd.h3.x = (-d.y*b.z + a.y*b.z - a.y*d.z + b.y*d.z + d.y*a.z - b.y*a.z)/denominator;
	sfd.h3.y = -(d.x*a.z - b.x*a.z - d.x*b.z + a.x*b.z - a.x*d.z + b.x*d.z)/denominator;
	sfd.h3.z = (-a.x*d.y + d.x*a.y - b.x*a.y - d.x*b.y + a.x*b.y + b.x*d.y)/denominator;

	sfd.h4.x = -(-c.z*a.y + a.y*b.z + b.y*c.z + c.y*a.z - b.y*a.z - c.y*b.z)/denominator;
	sfd.h4.y = (-a.x*c.z + c.x*a.z - b.x*a.z + b.x*c.z + a.x*b.z - c.x*b.z)/denominator;
	sfd.h4.z = -(-a.x*c.y - b.x*a.y + b.x*c.y + a.x*b.y - c.x*b.y + c.x*a.y)/denominator;

/*	printf("\nFor tetrahedron %i: \n", me_idx);
	printf("h1 derivatives: %f, %f, %f \n", sfd.h1.x, sfd.h1.y, sfd.h1.z);
	printf("h2 derivatives: %f, %f, %f \n", sfd.h2.x, sfd.h2.y, sfd.h2.z);
	printf("h3 derivatives: %f, %f, %f \n", sfd.h3.x, sfd.h3.y, sfd.h3.z);
	printf("h4 derivatives: %f, %f, %f \n", sfd.h4.x, sfd.h4.y, sfd.h4.z);
*/
	shape_function_derivatives[me_idx] = sfd;

}

__global__ void
precalculateABC(float4* ABCm, float* M, float timestep, float alpha, unsigned int numPoints)
{
	int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (me_idx>=numPoints)
		return;

	float twodelta = timestep*2.0f;
	float deltasqr = timestep*timestep;


	float Mii = M[me_idx];
	float Dii = alpha*Mii;  // mass-proportional damping is applied
	
//	printf("M: %f\n",Mii);

	float Ai = 1.0f/(Dii/twodelta + Mii/deltasqr);
	float Bi = ((2.0f*Mii)/deltasqr)*Ai;
	float Ci = (Dii/twodelta)*Ai - 0.5f*Bi;

//	printf("ABC for node %i: %f, %f, %f \n", me_idx, Ai, Bi, Ci);

	ABCm[me_idx] = make_float4(Ai,Bi,Ci,Mii);
}


__global__ void
updateDisplacements_k(float4 *Ui_t, float4 *Ui_tminusdt, float *M, float4 *Ri, float4 *Fi, int maxNumForces, float4 *ABC, unsigned int numPoints)
{
	int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (me_idx>=numPoints)
		return;

	float4 F = make_float4(0,0,0,0);

//	printf("Max num forces: %i\n", maxNumForces);

	for (int i=0; i<maxNumForces; i++)
	{
		float4 force_to_add = Fi[me_idx*maxNumForces+i];
		F.x += force_to_add.x;
		F.y += force_to_add.y;
		F.z += force_to_add.z;
	}
//	printf("Accumulated node %i force: %f, %f, %f \n", me_idx, F.x, F.y, F.z);

	float4 ABCi = ABC[me_idx];
	float4 Uit = Ui_t[me_idx];
	float4 Uitminusdt = Ui_tminusdt[me_idx];

	float4 R = Ri[me_idx];
	float x = ABCi.x * (R.x - F.x) + ABCi.y * Uit.x + ABCi.z * Uitminusdt.x;
	float y = ABCi.x * (R.y - F.y) + ABCi.y * Uit.y + ABCi.z * Uitminusdt.y;
	float z = ABCi.x * (R.z - F.z) + ABCi.y * Uit.z + ABCi.z * Uitminusdt.z;

/*	float x = ABCi.x * (-F.x) + ABCi.y * Ui_t[me_idx].x + ABCi.z * Ui_tminusdt[me_idx].x;
	float y = ABCi.x * (-F.x) + ABCi.y * Ui_t[me_idx].y + ABCi.z * Ui_tminusdt[me_idx].y;
	float z = ABCi.x * (-F.x ) + ABCi.y * Ui_t[me_idx].z + ABCi.z * Ui_tminusdt[me_idx].z;
*/
	Ui_tminusdt[me_idx] = make_float4(x,y,z,0);//XXXXXXXXXXXXXXXXXXXXX

}

struct Matrix4x3 //note: supposed to be castable to a ShapeFunctionDerivatives object
{
	float e[12];
};

struct Matrix3x3 
{
	float e[9];
};

struct Matrix6x3 
{
	float e[6*3];
};


texture<float4,  1, hipReadModeElementType> Ui_t_1d_tex;
texture<float,  1, hipReadModeElementType> V0_1d_tex;
texture<float4,  1, hipReadModeElementType> _tex;

#define h(i,j) (sfdm.e[(i-1)*3+(j-1)])
#define u(i,j) (displacements.e[(i-1)*3+(j-1)])
#define X(i,j) (deformation_gradients.e[(i-1)*3+(j-1)])
#define B(i,j) (b_tensor.e[(i-1)*3+(j-1)])
#define C(i,j) (cauchy_green_deformation.e[(i-1)*3+(j-1)])
#define CI(i,j) (c_inverted.e[(i-1)*3+(j-1)])
#define S(i,j) (s_tensor.e[(i-1)*3+(j-1)])

__global__ void
calculateForces_k(Matrix4x3 *shape_function_derivatives, Tetrahedron *tetrahedra, float4 *Ui_t, float *V_0, int4 *writeIndices, float4 *pointForces, int maxPointForces, float mu, float lambda, unsigned int numTets)
{
	int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (me_idx>=numTets)
		return;

	Tetrahedron e = tetrahedra[me_idx];

	if (e.x < 0) 
		return;
	
	Matrix4x3 sfdm = shape_function_derivatives[me_idx];
	Matrix4x3 displacements;

	//fill in displacement values in u (displacements)
	
	float3 U1 = crop_last_dim(tex1Dfetch( Ui_t_1d_tex, e.x ));
	float3 U2 = crop_last_dim(tex1Dfetch( Ui_t_1d_tex, e.y ));
	float3 U3 = crop_last_dim(tex1Dfetch( Ui_t_1d_tex, e.z ));
	float3 U4 = crop_last_dim(tex1Dfetch( Ui_t_1d_tex, e.w ));

	displacements.e[0] = U1.x;
	displacements.e[1] = U1.y;
	displacements.e[2] = U1.z;

	displacements.e[3] = U2.x;
	displacements.e[4] = U2.y;
	displacements.e[5] = U2.z;

	displacements.e[6] = U3.x;
	displacements.e[7] = U3.y;
	displacements.e[8] = U3.z;

	displacements.e[9] = U4.x;
	displacements.e[10] = U4.y;
	displacements.e[11] = U4.z;

	Matrix3x3 deformation_gradients;

	//Calculate deformation gradients
	X(1,1) = (u(1,1)*h(1,1)+u(2,1)*h(2,1)+u(3,1)*h(3,1)+u(4,1)*h(4,1)+1.0f); 
	X(1,2) = (u(1,1)*h(1,2)+u(2,1)*h(2,2)+u(3,1)*h(3,2)+u(4,1)*h(4,2));
	X(1,3) = (u(1,1)*h(1,3)+u(2,1)*h(2,3)+u(3,1)*h(3,3)+u(4,1)*h(4,3));

	X(2,1) = (u(1,2)*h(1,1)+u(2,2)*h(2,1)+u(3,2)*h(3,1)+u(4,2)*h(4,1));
	X(2,2) = (u(1,2)*h(1,2)+u(2,2)*h(2,2)+u(3,2)*h(3,2)+u(4,2)*h(4,2)+1.0f);
	X(2,3) = (u(1,2)*h(1,3)+u(2,2)*h(2,3)+u(3,2)*h(3,3)+u(4,2)*h(4,3));

	X(3,1) = (u(1,3)*h(1,1)+u(2,3)*h(2,1)+u(3,3)*h(3,1)+u(4,3)*h(4,1));
	X(3,2) = (u(1,3)*h(1,2)+u(2,3)*h(2,2)+u(3,3)*h(3,2)+u(4,3)*h(4,2));
	X(3,3) = (u(1,3)*h(1,3)+u(2,3)*h(2,3)+u(3,3)*h(3,3)+u(4,3)*h(4,3)+1.0f);

/*	printf("\nDeformation gradient tensor for tetrahedron %i: \n", me_idx);
	printf("%f, %f, %f \n", X(1,1), X(1,2), X(1,3));
	printf("%f, %f, %f \n", X(2,1), X(2,2), X(2,3));
	printf("%f, %f, %f \n", X(3,1), X(3,2), X(3,3));
*/
		// calculate Right Cauchy-Green deformation tensor C
		Matrix3x3 cauchy_green_deformation;

	C(1,1) = X(1, 1)*X(1, 1) + X(2, 1)*X(2, 1) + X(3, 1)*X(3, 1); 
	C(1,2) = X(1, 1)*X(1, 2) + X(2, 1)*X(2, 2) + X(3, 1)*X(3, 2); 
	C(1,3) = X(1, 1)*X(1, 3) + X(2, 1)*X(2, 3) + X(3, 1)*X(3, 3); 

	C(2,1) = X(1, 1)*X(1, 2) + X(2, 1)*X(2, 2) + X(3, 1)*X(3, 2); 
	C(2,2) = X(1, 2)*X(1, 2) + X(2, 2)*X(2, 2) + X(3, 2)*X(3, 2); 
	C(2,3) = X(1, 2)*X(1, 3) + X(2, 2)*X(2, 3) + X(3, 2)*X(3, 3);

	C(3,1) = X(1, 1)*X(1, 3) + X(2, 1)*X(2, 3) + X(3, 1)*X(3, 3); 
	C(3,2) = X(1, 2)*X(1, 3) + X(2, 2)*X(2, 3) + X(3, 2)*X(3, 3); 
	C(3,3) = X(1, 3)*X(1, 3) + X(2, 3)*X(2, 3) + X(3, 3)*X(3, 3);
/*
	printf("\nRight Cauchy-Green deformation tensor for tetrahedron %i: \n", me_idx);
	printf("%f, %f, %f \n", C(1,1), C(1,2), C(1,3));
	printf("%f, %f, %f \n", C(2,1), C(2,2), C(2,3));
	printf("%f, %f, %f \n", C(3,1), C(3,2), C(3,3));
*/

	//Invert C
	Matrix3x3 c_inverted;

	float denominator = (C(3, 1)*C(1, 2)*C(2, 3) - C(3, 1)*C(1, 3)*C(2, 2) - C(2, 1)*C(1, 2)*C(3, 3) 
		+ C(2, 1)*C(1, 3)*C(3, 2) + C(1, 1)*C(2, 2)*C(3, 3) - C(1, 1)*C(2, 3)*C(3, 2));

	CI(1,1) = (C(2, 2)*C(3, 3) - C(2, 3)*C(3, 2))/denominator; 
	CI(1,2) = (-C(1, 2)*C(3, 3) + C(1, 3)*C(3, 2))/denominator; 
	CI(1,3) = (C(1, 2)*C(2, 3) - C(1, 3)*C(2, 2))/denominator; 

	CI(2,1) = (-C(2, 1)*C(3, 3) + C(3, 1)*C(2, 3))/denominator; 
	CI(2,2) = (-C(3, 1)*C(1, 3) + C(1, 1)*C(3, 3))/denominator; 
	CI(2,3) = (-C(1, 1)*C(2, 3) + C(2, 1)*C(1, 3))/denominator; 

	CI(3,1) = (-C(3, 1)*C(2, 2) + C(2, 1)*C(3, 2))/denominator; 
	CI(3,2) = (-C(1, 1)*C(3, 2) + C(3, 1)*C(1, 2))/denominator; 
	CI(3,3) = (-C(2, 1)*C(1, 2) + C(1, 1)*C(2, 2))/denominator;

/*	printf("\nInverted right Cauchy-Green deformation tensor for tetrahedron %i: \n", me_idx);
	printf("%f, %f, %f \n", CI(1,1), CI(1,2), CI(1,3));
	printf("%f, %f, %f \n", CI(2,1), CI(2,2), CI(2,3));
	printf("%f, %f, %f \n", CI(3,1), CI(3,2), CI(3,3));
*/
	//Find the determinant of the deformation gradient
	float J = X(1, 1)*X(2, 2)*X(3, 3)-X(1, 1)*X(2, 3)*X(3, 2)+X(2, 1)*X(3, 2)*X(1, 3)-
		X(2, 1)*X(1, 2)*X(3, 3)+X(3, 1)*X(1, 2)*X(2, 3)-X(3, 1)*X(2, 2)*X(1, 3);

//	printf("\nDeterminant of the deformation gradient for tetrahedron %i: %f\n", me_idx, J);

	//Calculate stress tensor S from Neo-Hookean Model
	//  S(ij) = mu(delta(ij)-(C(ij)^(-1))^)+lambda^J(J-1)((C^(-1))(ij))

//	float mu = 1007.0f;
//	float lambda = 49329.0f;
	Matrix3x3 s_tensor;

	S(1,1) = mu*(1.0f-CI(1,1)) + lambda*J*(J-1.0f)*CI(1,1);
	S(2,2) = mu*(1.0f-CI(2,2)) + lambda*J*(J-1.0f)*CI(2,2); 
	S(3,3) = mu*(1.0f-CI(3,3)) + lambda*J*(J-1.0f)*CI(3,3);
	S(1,2) = mu*(-CI(1,2)) + lambda*J*(J-1.0f)*CI(1,2);
	S(2,3) = mu*(-CI(2,3)) + lambda*J*(J-1.0f)*CI(2,3);
	S(1,3) = mu*(-CI(1,3)) + lambda*J*(J-1.0f)*CI(1,3); // IS THIS RIGHT?? (3,1) instead?
//	S(1,3) = mu*(-CI(3,1)) + lambda*J*(J-1.0f)*CI(3,1); // IS THIS RIGHT?? (1,3) instead?


/*	printf("\nHyper-elastic stresses for tetrahedron %i: \n", me_idx);
	printf("%f, %f, %f \n", S(1,1), S(1,2), S(1,3));
	printf("%f, %f, %f \n", S(2,1), S(2,2), S(2,3));
	printf("%f, %f, %f \n", S(3,1), S(3,2), S(3,3));
*/
	float4 forces[4];

//	float V = V_0[me_idx];//look up volume
	float V = tex1Dfetch( V0_1d_tex, me_idx );

	//	printf("\nVolume for tetrahedron %i: %f\n", me_idx, V);

	for (int a=1; a<=4; a++) // all 4 nodes
	{
		//Calculate B_L from B_L0 and deformation gradients (a is the node number)

		Matrix6x3 b_tensor;

		B(1,1) = h(a, 1)*X(1, 1);  
		B(1,2) = h(a, 1)*X(2, 1);  
		B(1,3) = h(a, 1)*X(3, 1);  

		B(2,1) = h(a, 2)*X(1, 2);
		B(2,2) = h(a, 2)*X(2, 2);
		B(2,3) = h(a, 2)*X(3, 2);

		B(3,1) = h(a, 3)*X(1, 3);  
		B(3,2) = h(a, 3)*X(2, 3);  
		B(3,3) = h(a, 3)*X(3, 3);  

		B(4,1) = h(a, 2)*X(1, 1) + h(a, 1)*X(1, 2);  
		B(4,2) = h(a, 2)*X(2, 1) + h(a, 1)*X(2, 2);  
		B(4,3) = h(a, 2)*X(3, 1) + h(a, 1)*X(3, 2);  

		B(5,1) = h(a, 3)*X(1, 2) + h(a, 2)*X(1, 3);  
		B(5,2) = h(a, 3)*X(2, 2) + h(a, 2)*X(2, 3);  
		B(5,3) = h(a, 3)*X(3, 2) + h(a, 2)*X(3, 3);

		B(6,1) = h(a, 3)*X(1, 1) + h(a, 1)*X(1, 3);  
		B(6,2) = h(a, 3)*X(2, 1) + h(a, 1)*X(2, 3);  
		B(6,3) = h(a, 3)*X(3, 1) + h(a, 1)*X(3, 3);

/*		printf("\nSubmatrix for a=%i of the stationary strain-displacement matrix for tetrahedron %i: \n", a, me_idx);
		printf("%f, %f, %f \n", B(1,1), B(1,2), B(1,3));
		printf("%f, %f, %f \n", B(2,1), B(2,2), B(2,3));
		printf("%f, %f, %f \n", B(3,1), B(3,2), B(3,3));
		printf("%f, %f, %f \n", B(4,1), B(4,2), B(4,3));
		printf("%f, %f, %f \n", B(5,1), B(5,2), B(5,3));
		printf("%f, %f, %f \n", B(6,1), B(6,2), B(6,3));
*/
		//calculate forces
		float4 force;
		force.x = V*(B(1, 1)*S(1, 1)+B(2, 1)*S(2, 2)+B(3, 1)*S(3, 3)+B(4, 1)*S(1, 2)+B(5, 1)*S(2, 3)+B(6, 1)*S(1, 3));
		force.y = V*(B(1, 2)*S(1, 1)+B(2, 2)*S(2, 2)+B(3, 2)*S(3, 3)+B(4, 2)*S(1, 2)+B(5, 2)*S(2, 3)+B(6, 2)*S(1, 3));
		force.z = V*(B(1, 3)*S(1, 1)+B(2, 3)*S(2, 2)+B(3, 3)*S(3, 3)+B(4, 3)*S(1, 2)+B(5, 3)*S(2, 3)+B(6, 3)*S(1, 3));
		force.w = 0;

		if (length(crop_last_dim(force))<100000 && J>0)
			forces[a-1] = force;
		else
			forces[a-1] = make_float4(0,0,0,0);

	}

/*	printf("\nFor tetrahedron %i: \n", me_idx);
	printf("node1 (%i) force: %f, %f, %f \n", e.x, forces[0].x, forces[0].y, forces[0].z);
	printf("node2 (%i) force: %f, %f, %f \n", e.y, forces[1].x, forces[1].y, forces[1].z);
	printf("node3 (%i) force: %f, %f, %f \n", e.z, forces[2].x, forces[2].y, forces[2].z);
	printf("node4 (%i) force: %f, %f, %f \n", e.w, forces[3].x, forces[3].y, forces[3].z);
*/

	// look up where this tetrahedron is allowed to store its force contribution to a node
	// store force-vector
	pointForces[maxPointForces * e.x + writeIndices[me_idx].x] = forces[0];
	pointForces[maxPointForces * e.y + writeIndices[me_idx].y] = forces[1];
	pointForces[maxPointForces * e.z + writeIndices[me_idx].z] = forces[2];
	pointForces[maxPointForces * e.w + writeIndices[me_idx].w] = forces[3];

//	printf("Max num forces: %i\n", maxPointForces);

//	printf("%i, %i, %i, %i \n", writeIndices[me_idx].x, writeIndices[me_idx].y, writeIndices[me_idx].z, writeIndices[me_idx].w );
}

__global__ void
calculateDrivingForces_k(Point *points, float *masses, float4 *externalForces, unsigned int numPoints)
{
	int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (me_idx>=numPoints)
		return;

	externalForces[me_idx] = make_float4(0, -9820*masses[me_idx], 0, 0); // using millimeters - not meters - thus the factor 1000
//	externalForces[me_idx] = make_float4(0, 0, -9820*masses[me_idx], 0); // using millimeters - not meters - thus the factor 1000

}

__global__ void
applyGroundConstraint_k(Point *points, float4 *displacements, float4 *oldDisplacements,  float lowestYValue, unsigned int numPoints)
{
	int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (me_idx>=numPoints)
		return;

	Point me = points[me_idx];
	float4 displacement = displacements[me_idx];

	if ((me.y+displacement.y)<lowestYValue)
	{
		displacements[me_idx].y = lowestYValue - me.y;
		//oldDisplacements[me_idx] = displacements[me_idx];
	}
}
